#include "hip/hip_runtime.h"
#include "sph.cuh"
#include <hip/hip_fp16.h>

using namespace std;

void SPH_NS_simpleversion(float* sortedPos, float* sortedVel, float* sorteddensity, float* sortedpressure, int* sorted_particle_type, float* densitydt, float* Veldt, int* cellStart, int* cellEnd, int numParticles, int* particleHash, int timestep, float* dofv, float* rhop_sum, float* w_sum)
{
    int numThreads, numBlocks;
    //computeGridSize(numParticles, 128, numBlocks, numThreads); 
    computeGridSize(numParticles, 256, numBlocks, numThreads); 

    computeBoundary_Delta_acoustic_D<<<numBlocks,numThreads>>>(sortedPos, sortedVel, sorteddensity, sortedpressure, sorted_particle_type, cellStart, cellEnd, rhop_sum, w_sum, numParticles, particleHash, dofv);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        cout << "SPH_NS launch failed: " << hipGetErrorString(cudaStatus)<< __LINE__ << endl;
        system("pause");
    }

    computeGovering_equationD<<<numBlocks,numThreads>>>(sortedPos, sortedVel, sortedpressure, sorteddensity, sorted_particle_type, densitydt, dofv, Veldt, cellStart, cellEnd, numParticles, particleHash);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        cout << "SPH_NS launch failed: " << hipGetErrorString(cudaStatus)<< __LINE__ << endl;
        system("pause");
    }
}

__global__ void computeBoundary_Delta_acoustic_D(float* sortedPos, float* sortedVel, float* sorteddensity, float* sortedpressure, int* sorted_particle_type, int* cellStart, int* cellEnd, float* rhop_sum, float* w_sum, int numParticles, int* particleHash, float* dofv)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < numParticles)
    {
        //dofv[index] = 0.0; rhop_sum[index] = 0.0; w_sum[index] = 0.0;//dofv-acoustic damper
        float rhop_sum_tmp = 0.0f;
        float w_sum_tmp = 0.0f;
        float dofv_tmp = 0.0f;
        float3 pos,vel;
        pos.x = sortedPos[3 * index];
        pos.y = sortedPos[3 * index + 1];
        pos.z = sortedPos[3 * index + 2];
        vel.x = sortedVel[3 * index];
        vel.y = sortedVel[3 * index + 1];
        vel.z = sortedVel[3 * index + 2];
        int type_0 = sorted_particle_type[index];
        //float dens_0 = sorteddensity[index];
        //float pres_0 = sortedpressure[index];

        int type_1;
        float dens_1,pres_1;
        float dx,dy,dz;
        float rr,q,w,fr;
        float dvdx;
        __half half_dx;
        __half half_dy;
        __half half_dz;
        __half half_rr;

        for (int z = -3; z <= 3; z++)
        {
            for (int y = -3; y <= 3; y++)
            {

                    int newgridHash = particleHash[index] + z*par.gridxdim*par.gridydim + y*par.gridxdim;
                    if (newgridHash <= par.hash_max-3 && newgridHash >= 3)
                    {
                        //int startIndex = cellStart[newgridHash];
                        #define startIndex (cellStart[newgridHash-3])
                        //if (startIndex == 0xffffffff)	continue;
                        //int endIndex = cellEnd[newgridHash];
                        #define endIndex (cellEnd[newgridHash+3])
                        //  iterate over particles in this cell
                        for (int i = startIndex; i < endIndex; i++)
                        {
                            #undef startIndex
                            #undef endIndex
                            //int cellData = particleHash[i];
                            //if (cellData != newgridHash)  break;
                            if (i != index)	// check not colliding with self
                            {
                                dx = pos.x - sortedPos[3 * i];
                                dy = pos.y - sortedPos[3 * i + 1];
                                dz = pos.z - sortedPos[3 * i + 2];
                                half_dx = __float2half(dx);
                                half_dy = __float2half(dy);
                                half_dz = __float2half(dz);
                                half_rr = (half_dx*half_dx + half_dy*half_dy + half_dz*half_dz)/(__float2half(par.h)*__float2half(par.h));
                                //grid_count++;

                                if (half_rr <= __float2half(4.0f))
                                {
                                    pres_1 = sortedpressure[i];
                                    dens_1 = sorteddensity[i];
                                    type_1 = sorted_particle_type[i];

                                    rr = dx * dx + dy * dy + dz * dz;
                                    dvdx = (vel.x - sortedVel[3 * i])*dx + (vel.y - sortedVel[3 * i + 1])*dy + (vel.z - sortedVel[3 * i +2])*dz;
                                    q = sqrtf(rr)/par.h;
                                    fr = (1.0f - q/2.0f) * (1.0f - q/2.0f) * (1.0f - q/2.0f);
                                    w = fr*(1.0f - q/2.0f);
                                    w *= (2.0f*q + 1)*par.adh;
                                    //fr *= -5.0f*par.adh/(par.h*par.h);
                                    fr *= -5.0f*par.adh/(par.h2);
                                    
                                    if (type_0 != 1 && type_1 == 1)//计算边界所需变量
                                    {
                                        rhop_sum_tmp += (pres_1 - dens_1 * (0.0f * dx + 0.0f * dy + (0.0f - par.gravity) * dz)) * w;
                                        w_sum_tmp += w;
                                    }
                                    else if (type_0 == 1 && type_1 == 1)
                                    {
                                        dofv_tmp -= fr *dvdx * par.particleMass / dens_1;
                                    }
                                }
                            }
                        }
                    }
            }
        }
        dofv[index] = dofv_tmp;
        //if(par_count > 128) printf("the ptc :%d has %d pars and it's grid has %d ptcs!\n",index,par_count,grid_count);
        if (sorted_particle_type[index] != 1)
        {
            //if (fabs(w_sum[index]) > 1.0E-8)
            if (fabs(w_sum_tmp) > 1.0E-8f)
            {
                //sortedpressure[index] = rhop_sum[index] / w_sum[index];
                rhop_sum_tmp = rhop_sum_tmp / w_sum_tmp;
            }
            else
            {
                //sortedpressure[index] = 0;
                rhop_sum_tmp = 0.0f;
            }
            //if (sortedpressure[index] < 0)  sortedpressure[index] = 0;
            if(rhop_sum_tmp < 0.0f) rhop_sum_tmp = 0.0f;
            sortedpressure[index] = rhop_sum_tmp;
            sorteddensity[index] = rhop_sum_tmp /(par.cs2) + par.restDensity;
            //sorteddensity[index] = rhop_sum_tmp /(par.cs*par.cs) + par.restDensity;
        }
    }
}


__global__ void computeGovering_equationD(float* sortedPos, float* sortedVel, float* sortedpressure, float* sorteddensity, int* sorted_particle_type, float* densitydt, float* dofv, float* Veldt, int* cellStart, int* cellEnd, int numParticles, int* particleHash)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < numParticles)
    {
        float3 pos, vel;
        pos.x = sortedPos[3 * index];
        pos.y = sortedPos[3 * index + 1];
        pos.z = sortedPos[3 * index + 2];
        vel.x = sortedVel[3 * index];
        vel.y = sortedVel[3 * index + 1];
        vel.z = sortedVel[3 * index + 2];
        float pres_0 = sortedpressure[index];	
        float dens_0 = sorteddensity[index];
        float dofv_0 = dofv[index];
        int type_0 = sorted_particle_type[index];

        float dx,dy,dz;
        float dvdx;
        float pres_1,dens_1,dofv_1;
        //int type_1;

        float rr,fr;
        float q;
        float vtmp;

        float densitydt_temp = 0.0f;
        float3 veldt_temp = make_float3(0.0f, 0.0f, 0.0f);
        __half half_dx;
        __half half_dy;
        __half half_dz;
        __half half_rr;

        for (int z = -3; z <= 3; z++)
        {
            for (int y = -3; y <= 3; y++)
            {
                    int gridHash = particleHash[index] + z*par.gridxdim*par.gridydim + y*par.gridxdim ;

                    if (gridHash <= par.hash_max-3 && gridHash >= 3)
                    {
                        //int startIndex = cellStart[gridHash];
                        #define startIndex cellStart[gridHash-3]
                        //if (startIndex == 0xffffffff)	continue;
                        //int endIndex = cellEnd[gridHash];
                        #define endIndex cellEnd[gridHash+3]
                        //  iterate over particles in this cell
                        for (int i = startIndex; i < endIndex; i++)
                        {
                            #undef startIndex
                            #undef endIndex
                            //int cellData = particleHash[i];
                            //if (cellData != gridHash)  break;
                            if (i != index)	// check not colliding with self
                            {
                                dx = pos.x - sortedPos[3*i];
                                dy = pos.y - sortedPos[3*i+1];
                                dz = pos.z - sortedPos[3*i+2];

                                half_dx = __float2half(dx);
                                half_dy = __float2half(dy);
                                half_dz = __float2half(dz);
                                half_rr = (half_dx*half_dx + half_dy*half_dy + half_dz*half_dz)/(__float2half(par.h)*__float2half(par.h));
                                

                                if (half_rr <= __float2half(4.0f))
                                {
                                    dvdx = (vel.x - sortedVel[3*i])*dx + (vel.y - sortedVel[3*i+1])*dy + (vel.z - sortedVel[3*i+2])*dz;
                                    pres_1 = sortedpressure[i];
                                    dens_1 = sorteddensity[i];

                                    dofv_1 = dofv[i];
                                    //type_1 = sorted_particle_type[i];
                                
                                    rr = dx*dx + dy*dy + dz*dz;
                                    q = sqrtf(rr)/par.h;
                                    //fr = -5.0f * par.adh * (1.0f-q/2.0f) * (1.0f-q/2.0f) * (1.0f-q/2.0f)/(par.h*par.h);
                                    fr = -5.0f * par.adh * (1.0f-q/2.0f) * (1.0f-q/2.0f) * (1.0f-q/2.0f)/(par.h2);
                                    
                                    //densitydt_temp += (dens_0*dvdx*fr + (dens_0-dens_1)*rr*fr*par.delta*par.h*par.cs/(rr+par.eta*par.eta))*par.particleMass/dens_1;
                                    densitydt_temp += (dens_0*dvdx*fr + (dens_0-dens_1)*rr*fr*par.delta_h_cs/(rr+par.eta2))*par.particleMass/dens_1;
                                    vtmp = -(pres_0+pres_1)*par.particleMass*fr/(dens_0*dens_1);

                                    if (type_0 == 1 && sorted_particle_type[i] == 1)
                                    {
                                        vtmp += par.h_cs_rho_m*fr*(dofv_0+dofv_1+par.afa*dvdx/(rr+par.eta2))/(dens_0*dens_1);
                                        //vtmp += par.h*par.cs*par.restDensity*par.particleMass*fr*(dofv_0+dofv_1+par.afa*dvdx/(rr+par.eta*par.eta))/(dens_0*dens_1);
                                    }
                                    veldt_temp.x += vtmp*dx;
                                    veldt_temp.y += vtmp*dy;
                                    veldt_temp.z += vtmp*dz;
                                }
                            }
                        }
                    }
             
            }
        }
        __syncthreads();
        densitydt[index] = densitydt_temp;
        Veldt[3 * index] = veldt_temp.x;
        Veldt[3 * index + 1] = veldt_temp.y;
        Veldt[3 * index + 2] = veldt_temp.z + par.gravity;
    }
}
